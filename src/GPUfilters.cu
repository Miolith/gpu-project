#include "hip/hip_runtime.h"
#include <algorithm>
#include <assert.h>
#include <iostream>
#include <set>

#include "CImg.h"
#include "draw.h"
#include "tools.h"

using namespace std;
using namespace cimg_library;

__global__ void grayScaleKernel(rgba *image, int width, int height, size_t pitch)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < width && y < height)
    {
        image = (rgba *)((char *)image + y * pitch);
        int index = x;

        uint8_t gray =
            (image[index].red + image[index].green + image[index].blue) / 3;
        image[index].red = gray;
        image[index].green = gray;
        image[index].blue = gray;
    }
}

__constant__ float gfilter[3][3] = { { 1.0 / 16, 2.0 / 16, 1.0 / 16 },
                                     { 2.0 / 16, 4.0 / 16, 2.0 / 16 },
                                     { 1.0 / 16, 2.0 / 16, 1.0 / 16 } };

__global__ void gaussianBlurKernel(rgba *dst_image, rgba *src_image, int width,
                                   int height, size_t pitch)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < width && y < height)
    {
        dst_image = (rgba *)((char *)dst_image + y * pitch);

        int index = x;

        float red = 0.0f, green = 0.0f, blue = 0.0f;

        for (int i = -1; i <= 1; i++)
        {
            for (int j = -1; j <= 1; j++)
            {
                int x1 = x + j;
                int y1 = y + i;
                if (x1 >= 0 && x1 < width && y1 >= 0 && y1 < height)
                {
                    rgba* img = (rgba *)((char *)src_image + y1 * pitch);
                    int index1 = x1;
                    red += img[index1].red * gfilter[i + 1][j + 1];
                    green += img[index1].green * gfilter[i + 1][j + 1];
                    blue += img[index1].blue * gfilter[i + 1][j + 1];
                }
            }
        }

        dst_image[index].red = red;
        dst_image[index].green = green;
        dst_image[index].blue = blue;
    }
}

__device__ int my_abs(int value)
{
	return (value < 0) ? -value : value;
}

__global__ void imageDiffKernel(rgba *dst_image, rgba *ref, int width, int height, size_t pitch)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < width && y < height)
    {
        dst_image = (rgba *)((char *)dst_image + y * pitch);
        ref = (rgba *)((char *)ref + y * pitch);

        int index = x;

        int red = my_abs(dst_image[index].red - ref[index].red);
        int green = my_abs(dst_image[index].green - ref[index].green);
        int blue = my_abs(dst_image[index].blue - ref[index].blue);

        dst_image[index].red = red;
        dst_image[index].green = green;
        dst_image[index].blue = blue;
    }
}

__global__ void dilationKernel(rgba *image, bool **circleTable, rgba **refImg,
                               int width, int height, int precision)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < width && y < height)
    {
        int index = y * width + x;

        uint8_t maxi = 0;
        for (int yoffset = -precision; yoffset <= precision; yoffset++)
        {
            for (int xoffset = -precision; xoffset <= precision; xoffset++)
            {
                int new_y = y + yoffset;
                int new_x = x + xoffset;
                if (new_y < 0 || new_y >= height || new_x < 0 || new_x >= width
                    || circleTable[yoffset + precision][xoffset + precision])
                    continue;

                if (refImg[new_y][new_x].red > maxi)
                {
                    maxi = refImg[new_y][new_x].red;
                }
            }
        }
        image[index].red = maxi;
        image[index].green = maxi;
        image[index].blue = maxi;
    }
}

// call gray scale kernel
void grayScaleGPU(rgba *image, int width, int height)
{
    rgba *dst_image;
    size_t pitch;

    hipMallocPitch(&dst_image, &pitch, width * sizeof(rgba), height);
    hipMemcpy2D(dst_image, pitch, image, width * sizeof(rgba), width * sizeof(rgba),
                 height, hipMemcpyHostToDevice);
    int bsize = 32;
    int w = ceil((float)width / bsize);
    int h = ceil((float)height / bsize);

    dim3 threadsPerBlock(bsize, bsize);
    dim3 numBlocks(w, h);

    grayScaleKernel<<<numBlocks, threadsPerBlock>>>(dst_image, width, height, pitch);
    hipDeviceSynchronize();

    hipMemcpy2D(image, width * sizeof(rgba), dst_image, pitch, width * sizeof(rgba),
                height, hipMemcpyDeviceToHost);
    hipFree(dst_image);
}



void gaussianBlurGPU(rgba *image, int width, int height)
{
    rgba *dst_image, *src_image;
    size_t pitch;

    hipMallocPitch(&dst_image, &pitch, width * sizeof(rgba), height);
    hipMallocPitch(&src_image, &pitch, width * sizeof(rgba), height);
    
    hipMemcpy2D(dst_image, pitch, image, width * sizeof(rgba), width * sizeof(rgba),
                height, hipMemcpyHostToDevice);
    hipMemcpy2D(src_image, pitch, image, width * sizeof(rgba), width * sizeof(rgba),
                height, hipMemcpyHostToDevice);

    int bsize = 32;
    int w = ceil((float)width / bsize);
    int h = ceil((float)height / bsize);

    dim3 threadsPerBlock(bsize, bsize);
    dim3 numBlocks(w, h);

    gaussianBlurKernel<<<numBlocks, threadsPerBlock>>>(dst_image, src_image, width, height, pitch);
    hipDeviceSynchronize();

    hipMemcpy2D(image, width * sizeof(rgba), dst_image, pitch, width * sizeof(rgba),
                    height, hipMemcpyDeviceToHost);
    hipFree(dst_image);
    hipFree(src_image);
}

void imageDiffGPU(rgba *ref, rgba *image, int width, int height)
{
    rgba *dst_image, *ref_image;
    size_t pitch;

    hipMallocPitch(&dst_image, &pitch, width * sizeof(rgba), height);
    hipMallocPitch(&ref_image, &pitch, width * sizeof(rgba), height);

    hipMemcpy2D(dst_image, pitch, image, width * sizeof(rgba), width * sizeof(rgba),
                height, hipMemcpyHostToDevice);
    hipMemcpy2D(ref_image, pitch, ref, width * sizeof(rgba), width * sizeof(rgba),
                height, hipMemcpyHostToDevice);


    int bsize = 32;
    int w = ceil((float)width / bsize);
    int h = ceil((float)height / bsize);

    dim3 threadsPerBlock(bsize, bsize);
    dim3 numBlocks(w, h);

    imageDiffKernel<<<numBlocks, threadsPerBlock>>>(dst_image, ref_image, width, height, pitch);
    hipDeviceSynchronize();

    hipMemcpy2D(image, width * sizeof(rgba), dst_image, pitch, width * sizeof(rgba),
                height, hipMemcpyDeviceToHost);
    hipFree(dst_image);
}
void dilationGPU(rgba **image, int width, int height, int precision)
{
    bool **circleTable = getCircleTable(2 * precision);
    int line_size = width * sizeof(rgba);
    rgba *d_image;
    hipMalloc(&d_image, height * width * sizeof(rgba));
    for (int y = 0; y < height; y++)
    {
        hipMemcpy(d_image + y * width, image[y], line_size,
                   hipMemcpyHostToDevice);
    }
    dim3 threadsPerBlock(32, 32);
    dim3 numBlocks((width + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (height + threadsPerBlock.y - 1) / threadsPerBlock.y);

    dilationKernel<<<numBlocks, threadsPerBlock>>>(d_image, circleTable, image,
                                                   width, height, precision);
    for (int y = 0; y < height; y++)
    {
        hipMemcpy(image[y], d_image + y * width, line_size,
                   hipMemcpyDeviceToHost);
    }
    hipFree(d_image);
    freeCircleTable(circleTable, 2 * precision);
}
